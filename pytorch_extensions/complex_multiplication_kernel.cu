#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


template <typename scalar_t>
__device__ __forceinline__ void elementwise_operation(
        scalar_t a, scalar_t b,
        scalar_t c, scalar_t d,
        scalar_t* out_re, scalar_t* out_im
    ) {
    /****************************************************************
     * Complex Multiplication
     *
     * Operation (a + bj) * (c + dj)
     *
     * Result:   (a*c - b*d) + (a*d + b*c)j
     ****************************************************************/
    *out_re += a*c - b*d;
    *out_im += a*d + b*c;
}

template <typename scalar_t>
__global__ void complex_multiplication_cuda_kernel_v1(
        const torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits> x,
        const torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits> h,
        torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits> out,
        const int THREADS, const int C, const int W, const int PLANE_SIZE
    ){

    const int b = blockIdx.x; // Image position in Batch
    const int f = blockIdx.y; // Filter position

    const int cell_idx = blockIdx.z * THREADS + threadIdx.x; // data point/pixel/cell index in h x w plane
    if (cell_idx >= PLANE_SIZE) return;

    const int i = cell_idx/W;
    const int j = fmod(cell_idx, W);

    scalar_t out_re = 0.0;
    scalar_t out_im = 0.0;

    /****************************************************************
     * Dimensions should be 
     * x   -> (B, C, H, W, 2) ~> Each b is size C*H*W*I
     * h   -> (F, C, H, W, 2) ~>      f is  ""
     * out -> (B, F, H, W, 2) ~>      b is  ""  F*H*W*I
     ****************************************************************/
    for (int c = 0; c < C; ++c) {

        const scalar_t x_re = x[b][c][i][j][0];
        const scalar_t x_im = x[b][c][i][j][1];

        const scalar_t h_re = h[f][c][i][j][0];
        const scalar_t h_im = h[f][c][i][j][1];

        elementwise_operation(x_re, x_im, h_re, h_im, &out_re, &out_im);
    }

    out[b][f][i][j][0] = out_re;
    out[b][f][i][j][1] = out_im;
}

/**
 * Multiplies two tensors of Complex Tensors
 * @param x
 * @param h
 * @param output
 */
at::Tensor complex_multiplication_cuda_v1(at::Tensor x, at::Tensor h) {
    const int THREADS = 1024;

    const int B = x.size(0);
    const int F = h.size(0);
    const int C = x.size(1);
    const int H = x.size(2);
    const int W = x.size(3);
    const int PLANE_SIZE = H*W;

    const auto Z = (H*W + THREADS - 1)/THREADS;
    const dim3 GRID_SIZE(B, F, Z);

    auto output = torch::zeros(
        {B, F, H, W, 2},
        torch::TensorOptions().device(x.device().type(), x.device().index())
        );

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "complex_multiplication_cuda_v1",
        ([&] {
            complex_multiplication_cuda_kernel_v1<scalar_t><<<GRID_SIZE, THREADS>>>(
                x.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>(),
                h.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>(),
                output.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>(),
                THREADS, C, W, PLANE_SIZE
            );
        })
    );

    return output;
}
